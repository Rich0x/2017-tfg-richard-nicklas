#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define SIZE 1024

//CUDA enabled vectorAdd
__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	//int i;

	int i = threadIdx.x;

	//for (i = 0; i < n; ++i)
	if(i<n)

		c[i] = a[i] + b[i];
}

//Normal vectorADD
void VectorAdd(int *a, int *b, int *c, int n)
{
	int i;

	for (i = 0; i < n; ++i)

		c[i] = a[i] + b[i];
}

int main()
{


	int *a, *b, *c, *d;
	int *d_a, *d_b, *d_c;

	//allocate CPU
	a = (int *)malloc(SIZE*sizeof(int));
	b = (int *)malloc(SIZE*sizeof(int));
	c = (int *)malloc(SIZE*sizeof(int));
	d = (int *)malloc(SIZE*sizeof(int));

	//allocate GPU
	hipMalloc(&d_a, SIZE*sizeof(int));
	hipMalloc(&d_b, SIZE*sizeof(int));
	hipMalloc(&d_c, SIZE*sizeof(int));

	//initialize CPU
	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	//initialize GPU
	hipMemcpy(d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice);


	//<<<NOfBlocks,ThreadsPerBlock>>>
	VectorAdd<<<1, SIZE>>>(d_a,d_b,d_c, SIZE);

	VectorAdd (a, b, c, SIZE);

	//recover result from GPU memory
	hipMemcpy(d, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; i++)
		printf("d[%d] =%d\n", i,d[i]);

	//free mem
	free(a);
	free(b);
	free(c);
	free(d);

	//free CUDA mem
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
