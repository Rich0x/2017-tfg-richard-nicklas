#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime_api.h>
#define SIZE 1024

//CUDA enabled vectorAdd
__global__ void vectorAdd(int *a, int *b, int *c, int n)
{
	//int i;

	int i = threadIdx.x;

	//for (i = 0; i < n; ++i)
	if(i<n)

		c[i] = a[i] + b[i];
}

//Normal vectorADD
void classicVectorAdd(int *a, int *b, int *c, int n)
{
	int i;

	for (i = 0; i < n; ++i)

		c[i] = a[i] + b[i];
}

int main()
{


	int *a, *b, *c, *d;
	int *d_a, *d_b, *d_c;

	//allocate CPU
	a = (int *)malloc(SIZE*sizeof(int));
	b = (int *)malloc(SIZE*sizeof(int));
	c = (int *)malloc(SIZE*sizeof(int));
	d = (int *)malloc(SIZE*sizeof(int));

	//allocate GPU
	checkCudaErrors(hipMalloc(&d_a, SIZE*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_b, SIZE*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_c, SIZE*sizeof(int)));

	//initialize CPU
	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	//initialize GPU
	checkCudaErrors(hipMemcpy(d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice));


	//<<<NOfBlocks,ThreadsPerBlock>>>
	vectorAdd<<<1, SIZE>>>(d_a,d_b,d_c, SIZE);

	classicVectorAdd (a, b, c, SIZE);

	//recover result from GPU memory
	hipMemcpy(d, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; i++)
		printf("d[%d] =%d\n", i,d[i]);

	//free mem
	free(a);
	free(b);
	free(c);
	free(d);

	//free CUDA mem
	checkCudaErrors(hipFree(d_a));
	checkCudaErrors(hipFree(d_b));
	checkCudaErrors(hipFree(d_c));

	return 0;
}
