

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 1024

//se ejecuta en GPU
__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	//int i;

	int i = threadIdx.x;

	//for (i = 0; i < n; ++i)
	if(i<n)

		c[i] = a[i] + b[i];
}

void OldVectorAdd(int *a, int *b, int *c, int n)
{
	int i;

	for (i = 0; i < n; ++i)

		c[i] = a[i] + b[i];
}

int main()
{


	int *a, *b, *c, *d;
	int *d_a, *d_b, *d_c;

	//allocate CPU/GPU
	a = (int *)malloc(SIZE*sizeof(int));
	b = (int *)malloc(SIZE*sizeof(int));
	c = (int *)malloc(SIZE*sizeof(int));
	d = (int *)malloc(SIZE*sizeof(int));

	hipMalloc(&d_a, SIZE*sizeof(int));
	hipMalloc(&d_b, SIZE*sizeof(int));
	hipMalloc(&d_c, SIZE*sizeof(int));

	//initialize CPU
	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	//initialize GPU
	hipMemcpy(d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice);

	//VectorAdd(a,b,c, SIZE);
	//<<<NOfBlocks,ThreadsPerBlock>>>
	VectorAdd<<<1, SIZE>>>(d_a,d_b,d_c, SIZE);

	OldVectorAdd (a, b, c, SIZE);


	hipMemcpy(d, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; i++)
		printf("d[%d] =%d\n", i,d[i]);

	free(a);
	free(b);
	free(c);
	free(d);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
